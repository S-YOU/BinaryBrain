#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "cubb/MicroMlp.h"


#define CUDA_SAFE_CALL(func) \
do { \
     hipError_t err = (func); \
     if (err != hipSuccess) { \
         fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
         exit(1); \
     } \
} while(0)



#if 1


//#define MAX_NODE_SIZE	2048
#define	N				6
#define	M				16

//__constant__ int g_input_index[MAX_NODE_SIZE*N];


__global__ void kernal_MicroMlp6x16_forward(
			int				frame_size,
			const float*	in_sig,
			float*			out_sig,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	int frame_step = blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
				 float b1;

	// �W���ǂݍ���
	for ( int i = 0; i < M; ++i ) {
		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}
	}
	for ( int i = 0; i < M; ++i ) {
		b0[i] = hidden_b[node * M + i];
	}
	for ( int i = 0; i < M; ++i ) {
		W1[i] = output_W[node * M + i];
	}
	b1 = output_b[node];
	
	__syncthreads();

	const float *in_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_ptr[i] = &in_sig[frame_size * in_idx];
	}

	float *out_ptr = &out_sig[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_data[N];
		for ( int i = 0; i < N; ++i ) {
			in_data[i] = in_ptr[i][frame];
		}

		// �v�Z
		float acc1 = b1;
		for ( int i = 0; i < M; ++i ) {
			float acc0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				acc0 += in_data[j] * W0[i][j];
			}
		
			acc0 = fmaxf(acc0, 0);	// ReLU
		
			acc1 += acc0 * W1[i];
		}

		// �o��
		out_ptr[frame] = acc1;

		frame += frame_step;
	}
}


int MicroMlp6x16_Forward
		(
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const float*	in_sig,
			float*			out_sig,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	hipDeviceProp_t dev;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig;
	float* dev_out_sig;
	int*   dev_input_index;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;

	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig,   input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_sig,  output_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

//	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_input_index), input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(dev_input_index, input_index, output_node_size * N * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_in_sig, in_sig, input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();
	
	dim3	grid(output_node_size);
	dim3	block(128*4, 1, 1);
	
	kernal_MicroMlp6x16_forward<<<grid, block>>>(
			frame_size,
			dev_in_sig,
			dev_out_sig,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
    }


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(out_sig, dev_out_sig, output_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipFree(dev_in_sig));
	CUDA_SAFE_CALL(hipFree(dev_out_sig));
	CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	CUDA_SAFE_CALL(hipFree(dev_output_W));
	CUDA_SAFE_CALL(hipFree(dev_output_b));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();
	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]" << std::endl;
	double flops = (double)output_node_size *(double) frame_size * (16.0*6.0+16.0+16.0)*2.0 / elapsed_kernel / 1000000.0;
	std::cout << "      " << flops << " [GFLOPS]  (" << flops / 942.0 * 100.0 << "% [peak 942 GFLOPS])" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}



#elif 1

#define MAX_NODE_SIZE	512
#define MAX_FRAME_UNIT	64
#define	N				6
#define	M				16

__constant__ int g_input_index[MAX_NODE_SIZE*N];


__global__ void kernal_MicroMlp6x16_forward(
			const float*	in_sig,
			float*			out_sig,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	int frame_size = gridDim.x * blockDim.x;
	int frame = blockIdx.x * blockDim.x + threadIdx.x;
	int node  = blockIdx.y * blockDim.y + threadIdx.y;
	
	// ���̓f�[�^�ǂݍ���
	float	in_data[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = g_input_index[node*N + i];
		in_data[i] = in_sig[frame_size * in_idx + frame];
	}

	// ���i�v�Z	
	float	hidden_data[M];
	for ( int i = 0; i < M; ++i ) {
		float acc = hidden_b[node * M + i];
		for ( int j = 0; j < N; ++j ) {
			acc += in_data[j] * hidden_W[(node * M + i) * N + j];
		}
		
		acc = fmaxf(acc, 0);	// ReLU
		
		hidden_data[i] = acc;
	}

	// �o�͒i�v�Z
	{
		float acc = output_b[node];
		for ( int i = 0; i < M; ++i ) {
			acc += hidden_data[i] * output_W[node * M + i];
		}

		out_sig[frame_size * node + frame] = acc;
	}
}


int MicroMlp6x16_Forward
		(
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const float*	in_sig,
			float*			out_sig,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	hipDeviceProp_t dev;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig;
	float* dev_out_sig;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;

	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig,   input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_sig,  output_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_input_index), input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_in_sig, in_sig, input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();
	
	dim3	grid((frame_size + 31) / 32, (output_node_size + 31) / 32);
	dim3	block(frame_size / grid.x, output_node_size / grid.y);
	
	kernal_MicroMlp6x16_forward<<<grid, block>>>(
			dev_in_sig,
			dev_out_sig,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
    }


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(out_sig, dev_out_sig, output_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipFree(dev_in_sig));
	CUDA_SAFE_CALL(hipFree(dev_out_sig));
	CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	CUDA_SAFE_CALL(hipFree(dev_output_W));
	CUDA_SAFE_CALL(hipFree(dev_output_b));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();
	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]" << std::endl;
	double flops = (double)output_node_size *(double) frame_size * (16.0*6.0+16.0+16.0)*2.0 / elapsed_kernel / 1000000.0;
	std::cout << "      " << flops << " [GFLOPS]  (" << flops / 942.0 * 100.0 << "% [peak 942 GFLOPS])" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}


#else

#define MAX_NODE_SIZE	512
#define MAX_FRAME_UNIT	64
#define	N				6
#define	M				16

__constant__ int g_input_index[MAX_NODE_SIZE*N];

__global__ void kernal_MicroMlp6x16_forward(
			const float*	in_sig,
			float*			out_sig,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	__shared__   float hidden_buf[M*MAX_FRAME_UNIT];
	__shared__   float input_buf [N*MAX_FRAME_UNIT];

//	int frame_size = gridDim.z * gridDim.x * blockDim.x;
//	int frame = (blockDim.z * blockIdx.z + blockIdx.x) * blockDim.x + threadIdx.x;
	int frame_size = gridDim.x * blockDim.x;
	int frame = (blockIdx.x) * blockDim.x + threadIdx.x;
	int index = threadIdx.y;
	int node  = blockIdx.y;

	// input_buf(shared memory) �� �R�s�[
	if ( index < N ) {
		int in_idx = g_input_index[node*N + index];
		input_buf[index * MAX_FRAME_UNIT + threadIdx.x] = in_sig[frame_size * in_idx + frame];
	}

	__syncthreads();

	const float*	ptr_in;
	float*			ptr_out;
	const float*	ptr_W;
	const float*	ptr_b;
	float			acc;

	// ���i�v�Z
	ptr_in  = &input_buf[threadIdx.x];
	ptr_W = &hidden_W[(node * M + index) * N];
	ptr_b = &hidden_b[node * M + index];
	acc = ptr_b[0];
	for ( int i = 0; i < N; ++i ) {
		acc += ptr_in[i * MAX_FRAME_UNIT] * ptr_W[i];
	}
	acc = fmaxf(acc, 0);	// ReLU

	ptr_out = &hidden_buf[threadIdx.x];
	ptr_out[index * MAX_FRAME_UNIT] = acc;

	__syncthreads();

	// �o�͒i�v�Z
	if ( index == 0 ) {
		ptr_in = ptr_out;
		ptr_W = &output_W[node * M];
		ptr_b = &output_b[node];
		acc = ptr_b[0];
		for ( int i = 0; i < M; ++i ) {
			acc += ptr_in[i * MAX_FRAME_UNIT] * ptr_W[i];
		}

		ptr_out = &out_sig[frame];
		ptr_out[frame_size * node] = acc;
	}
}


int MicroMlp6x16_Forward
		(
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const float*	in_sig,
			float*			out_sig,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	hipDeviceProp_t dev;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig;
	float* dev_out_sig;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;

	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig,   input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_sig,  output_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	
//	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_input_index), input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

//	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_in_sig, in_sig, input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

//	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();

	int		frame_unit = frame_size;
	if ( frame_unit > MAX_FRAME_UNIT ) { frame_unit = MAX_FRAME_UNIT; }

	int grid_x = frame_size / frame_unit;
	int grid_z = 1; 
//	while (grid_x > 1024) {
//		grid_x /= 2;
//		grid_z *= 2;
//	}
	dim3	grid(grid_x, output_node_size, grid_z);
	dim3	block(frame_unit, M);

	kernal_MicroMlp6x16_forward<<<grid, block>>>(
			dev_in_sig,
			dev_out_sig,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
    }


//	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(out_sig, dev_out_sig, output_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

//	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipFree(dev_in_sig));
	CUDA_SAFE_CALL(hipFree(dev_out_sig));
	CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	CUDA_SAFE_CALL(hipFree(dev_output_W));
	CUDA_SAFE_CALL(hipFree(dev_output_b));

//	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();
	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;

	return 0;
}


#endif

