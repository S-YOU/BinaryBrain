#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu_util.h"



// -------------------------------------------------
//  Forward
// -------------------------------------------------

template <int N=6, int M=16>
__global__ void kernal_MicroMlp_forward(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b)
{
	int frame_step = blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
	__shared__	 float b1;

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
	if (threadIdx.x == 0) {
		b1 = output_b[node];
	}

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float *out_sig_ptr = &out_sig_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}

		// �v�Z
		float sig1 = b1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU
		
			sig1 += sig0 * W1[i];
		}

		// �o��
		out_sig_ptr[frame] = sig1;

		frame += frame_step;
	}
}


template <int N=6, int M=16>
int bbcu_MicroMlp_Forward
		(
			const float*	dev_in_sig,
			float*			dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId
		)
{
	dim3	grid(output_node_size);
	dim3	block(512, 1, 1);
	
	kernal_MicroMlp_forward<N, M><<<grid, block, 0, streamId>>>(
			dev_in_sig,
			dev_out_sig,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);

	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 1;
    }
	
	return 0;
}


int bbcu_MicroMlp6x16_Forward
		(
			const float*	dev_in_sig,
			float*			dev_out_sig,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			const float*	dev_output_W,
			const float*	dev_output_b,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_Forward<6, 16>(
			dev_in_sig,
			dev_out_sig,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b,
			streamId
		);
}



template <int N=6, int M=16>
int MicroMlp_Forward
		(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	hipDeviceProp_t dev;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig;
	float* dev_out_sig;
	int*   dev_input_index;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;

	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig,   input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_sig,  output_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

//	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g_input_index), input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(dev_input_index, input_index, output_node_size * N * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_in_sig, in_sig_buf, input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();
	
	dim3	grid(output_node_size);
	dim3	block(128*4, 1, 1);
	
	kernal_MicroMlp_forward<<<grid, block, 0, 0>>>(
			dev_in_sig,
			dev_out_sig,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_output_W,
			dev_output_b
		);
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(1);
    }


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(out_sig_buf, dev_out_sig, output_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipFree(dev_in_sig));
	CUDA_SAFE_CALL(hipFree(dev_out_sig));
	CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	CUDA_SAFE_CALL(hipFree(dev_output_W));
	CUDA_SAFE_CALL(hipFree(dev_output_b));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();

	double kernel_flops = (double)output_node_size *(double) frame_size * (M*N+M+M)*2.0 / elapsed_kernel / 1000000.0;

	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]  " << kernel_flops << " [GFLOPS]" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}


int MicroMlp6x16_Forward
		(
			const float*	in_sig_buf,
			float*			out_sig_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			const float*	output_W,
			const float*	output_b
		)
{
	return MicroMlp_Forward
		(
			in_sig_buf,
			out_sig_buf,
			input_node_size,
			output_node_size,
			frame_size,
			input_index,
			hidden_W,
			hidden_b,
			output_W,
			output_b
		);
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------


#if 1


template <int N=6, int M=16, int H=16>
__global__ void kernal_MicroMlp_backward(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db)
{
	int	id         = threadIdx.x;
	int frame_step = H;	// blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
//				 float b1;

 	__shared__   float dW0[M][N][H];
	__shared__   float db0[M][H];
	__shared__   float dW1[M][H];
	__shared__	 float db1[H];

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
//	if (threadIdx.x == 0) {
//		b1 = output_b[node];
//	}
	
	// ���z������
	for ( int i = 0; i < M; ++ i ) {
		for ( int j = 0; j < N; ++j ) {
			dW0[i][j][id] = 0; // hidden_dW[(node * M + i) * N + j];
		}
	}
	for ( int i = 0; i < M; ++i ) {
		db0[i][id] = 0; // hidden_db[node * M + i];
	}
	for ( int i = 0; i < M; ++i ) {
		dW1[i][id] = 0; // output_dW[node * M + i];
	}
	db1[id] = 0; // output_db[node];

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float	*out_err_ptr = &out_err_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}
		
		// 1�i�ڍČv�Z����2�i�ڋt�`�d
		float	err1 = out_err_ptr[frame];
		float	err0[M];
		db1[id] += err1;
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU

			dW1[i][id] += err1 * sig0;

			if ( sig0 > 0 ) {		// ReLU
				err0[i] = err1 * W1[i];
			}
			else {
				err0[i] = 0;
			}
		}
		
		// 1�i�ڋt�`�d
		float *in_err_ptr  = &in_err_buf[frame_size * N * node];
		float	in_err[N];
		for ( int i = 0; i < N; ++i ) {
			in_err[i] = 0;	// in_err_ptr[frame_size * i + frame];
		}

		for ( int i = 0; i < M; ++i ) {
			db0[i][id] += err0[i];
			for ( int j = 0; j < N; ++j ) {
				dW0[i][j][id] += err0[i] * in_sig[j];
				in_err[j] += err0[i] * W0[i][j];
			}
		}
		
		// �덷��������
		for ( int i = 0; i < N; ++i ) {
			in_err_ptr[frame_size * i + frame] = in_err[i];
		}

		frame += frame_step;
	}
	
	__syncthreads();

	int comb = 1;
	while ( comb < H ) {
		int next = comb * 2;
		int mask = next - 1;
		if ( (threadIdx.x & mask) == 0 && id + comb < H ) {
			for ( int i = 0; i < M; ++ i ) {
				for ( int j = 0; j < N; ++j ) {
					dW0[i][j][id] += dW0[i][j][id + comb];
				}
			}
			for ( int i = 0; i < M; ++i ) {
				db0[i][id] += db0[i][id + comb];
			}
			for ( int i = 0; i < M; ++i ) {
				dW1[i][id] += dW1[i][id + comb];
			}
			db1[id] += db1[id + comb];
		}
		comb = next;
		__syncthreads();
	}

	// ���z�o��(��ŕ��񉻂���)
	if ( threadIdx.x == 0 ) {
		for ( int i = 0; i < M; ++i ) {
			for ( int j = 0; j < N; ++j ) {
				hidden_dW[(node * M + i) * N + j] = dW0[i][j][0];
			}
		}
		for ( int i = 0; i < M; ++i ) {
			hidden_db[node * M + i] = db0[i][0];
		}
		for ( int i = 0; i < M; ++i ) {
			output_dW[node * M + i] = dW1[i][0];
		}
		output_db[node] = db1[0];
	}
}




#else

template <int N=6, int M=16>
__global__ void kernal_MicroMlp_backward(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db)
{
	int frame_step = blockDim.x;
	int frame      = threadIdx.x;
	int node       = blockIdx.x;

	__shared__   float W0[M][N];
	__shared__   float b0[M];
	__shared__   float W1[M];
//				 float b1;

 	__shared__   float dW0[M][N];
	__shared__   float db0[M];
	__shared__   float dW1[M];
	__shared__	 float db1;

	// �W���ǂݍ���
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			W0[i][j] = hidden_W[(node * M + i) * N + j];
		}

		b0[i] = hidden_b[node * M + i];
		W1[i] = output_W[node * M + i];
	}
//	if (threadIdx.x == 0) {
//		b1 = output_b[node];
//	}
	
	// ���z������
	if (threadIdx.x < M) {
		int i = threadIdx.x;

		for ( int j = 0; j < N; ++j ) {
			dW0[i][j] = 0; // hidden_dW[(node * M + i) * N + j];
		}
		db0[i] = 0; // hidden_db[node * M + i];
		dW1[i] = 0; // output_dW[node * M + i];
	}
	if (threadIdx.x == 0) {
		db1 = 0; // output_db[node];
	}

	__syncthreads();

	const float *in_sig_ptr[N];
	for ( int i = 0; i < N; ++i ) {
		int in_idx = input_index[node*N + i];
		in_sig_ptr[i] = &in_sig_buf[frame_size * in_idx];
	}

	float	*out_err_ptr = &out_err_buf[frame_size * node];

	// 1��SM��1node��S�t���[������
	while ( frame <  frame_size ) {
		// ���̓f�[�^�ǂݍ���
		float	in_sig[N];
		for ( int i = 0; i < N; ++i ) {
			in_sig[i] = in_sig_ptr[i][frame];
		}
		
		// 1�i�ڍČv�Z����2�i�ڋt�`�d
		float	err1 = out_err_ptr[frame];
		float	err0[M];
		atomicAdd(&db1, err1);
		for ( int i = 0; i < M; ++i ) {
			float sig0 = b0[i];
			for ( int j = 0; j < N; ++j ) {
				sig0 += in_sig[j] * W0[i][j];
			}
		
			sig0 = fmaxf(sig0, 0);	// ReLU

			atomicAdd(&dW1[i], err1 * sig0);

			if ( sig0 > 0 ) {		// ReLU
				err0[i] = err1 * W1[i];
			}
			else {
				err0[i] = 0;
			}
		}
		
		// 1�i�ڋt�`�d
		float *in_err_ptr  = &in_err_buf[frame_size * N * node];
		float	in_err[N];
		for ( int i = 0; i < N; ++i ) {
			in_err[i] = 0;	// in_err_ptr[frame_size * i + frame];
		}

		for ( int i = 0; i < M; ++i ) {
			atomicAdd(&db0[i], err0[i]);
			for ( int j = 0; j < N; ++j ) {
				atomicAdd(&dW0[i][j], err0[i] * in_sig[j]);
				in_err[j] += err0[i] * W0[i][j];
			}
		}
		
		// �덷��������
		for ( int i = 0; i < N; ++i ) {
			in_err_ptr[frame_size * i + frame] = in_err[i];
		}

		frame += frame_step;
	}
	
	__syncthreads();

	// ���z�o��(��ŕ��񉻂���)
	if ( threadIdx.x == 0 ) {
		for ( int i = 0; i < M; ++i ) {
			for ( int j = 0; j < N; ++j ) {
				hidden_dW[(node * M + i) * N + j] = dW0[i][j];
			}
		}
		for ( int i = 0; i < M; ++i ) {
			hidden_db[node * M + i] = db0[i];
		}
		for ( int i = 0; i < M; ++i ) {
			output_dW[node * M + i] = dW1[i];
		}
		output_db[node] = db1;
	}
}

#endif


template <int N=6>
__global__ void kernal_MicroMlp_backward_marge(
			float*			dst_buf,
			const float*	src_buf,
			int				frame_size,
			int				node_size,
			const int*		input_index
		)
{
	int n          = blockDim.y * blockIdx.y + threadIdx.y;
	int frame      = blockDim.x * blockIdx.x + threadIdx.x;
	
	for ( int node = 0; node < node_size; ++node ) {
		int in_idx = input_index[node*N + n];
		float*		 dst_buf_ptr = &dst_buf[frame_size * in_idx];
		const float* src_buf_ptr = &src_buf[(N * node + n) * frame_size];
		
		dst_buf_ptr[frame] += src_buf_ptr[frame];

		__syncthreads();
	}
}


#if 0
template <int N=6, int M=16>
int bbcu_MicroMlp_backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId = 0
	)
{
	{
		dim3	grid(output_node_size);
		dim3	block(256, 1, 1);
		
		kernal_MicroMlp_backward<N, M><<<grid, block, 0, streamId>>>(
				dev_in_sig_buf,
				dev_in_err_tmp,
				dev_out_err_buf,
				frame_size,
				dev_input_index,
				dev_hidden_W,
				dev_hidden_b,
				dev_hidden_dW,
				dev_hidden_db,
				dev_output_W,
				dev_output_b,
				dev_output_dW,
				dev_output_db
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	{
		int block_x = frame_size;
		while ( block_x > 1024 ) { block_x /= 2; }

		dim3	grid(frame_size/block_x, N);
		dim3	block(block_x, 1, 1);

		kernal_MicroMlp_backward_marge<N><<<grid, block>>>(
				dev_in_err_buf,
				dev_in_err_tmp,
				frame_size,
				output_node_size,
				dev_input_index
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	return 0;
}
#endif


template <int N=6, int M=16>
int bbcu_MicroMlp_backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId = 0
	)
{
	{
		const int x_size = (8192 / (N*M));

		dim3	grid(output_node_size);
		dim3	block(x_size, 1, 1);
		
		kernal_MicroMlp_backward<N, M, x_size><<<grid, block, 0, streamId>>>(
				dev_in_sig_buf,
				dev_in_err_tmp,
				dev_out_err_buf,
				frame_size,
				dev_input_index,
				dev_hidden_W,
				dev_hidden_b,
				dev_hidden_dW,
				dev_hidden_db,
				dev_output_W,
				dev_output_b,
				dev_output_dW,
				dev_output_db
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	{
		int block_x = frame_size;
		while ( block_x > 1024 ) { block_x /= 2; }

		dim3	grid(frame_size/block_x, N);
		dim3	block(block_x, 1, 1);

		kernal_MicroMlp_backward_marge<N><<<grid, block>>>(
				dev_in_err_buf,
				dev_in_err_tmp,
				frame_size,
				output_node_size,
				dev_input_index
			);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return 1;
		}
	}

	return 0;
}


int bbcu_MicroMlp6x16_backward(
			const float*	dev_in_sig_buf,
			float*			dev_in_err_buf,
			float*			dev_in_err_tmp,
			float*			dev_out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		dev_input_index,
			const float*	dev_hidden_W,
			const float*	dev_hidden_b,
			float*			dev_hidden_dW,
			float*			dev_hidden_db,
			const float*	dev_output_W,
			const float*	dev_output_b,
			float*			dev_output_dW,
			float*			dev_output_db,
			hipStream_t	streamId
		)
{
	return bbcu_MicroMlp_backward<6, 16>(
			dev_in_sig_buf,
			dev_in_err_buf,
			dev_in_err_tmp,
			dev_out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_hidden_dW,
			dev_hidden_db,
			dev_output_W,
			dev_output_b,
			dev_output_dW,
			dev_output_db,
			streamId
		);
}



template <int N=6, int M=16>
int MicroMlp_Backward
		(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db
		)
{
	hipDeviceProp_t dev;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&dev, 0));

	hipError_t cudaStatus0 = hipGetLastError();
    if (cudaStatus0 != hipSuccess) {
        fprintf(stderr, "start failed: %s\n", hipGetErrorString(cudaStatus0));
		exit(1);
    }

	hipDeviceSynchronize();
	auto time0 = std::chrono::system_clock::now();

	float* dev_in_sig_buf;
	float* dev_in_err_buf;
	float* dev_in_err_tmp;
	float* dev_out_err_buf;

	int*   dev_input_index;
	float* dev_hidden_W;
	float* dev_hidden_b;
	float* dev_output_W;
	float* dev_output_b;
	float* dev_hidden_dW;
	float* dev_hidden_db;
	float* dev_output_dW;
	float* dev_output_db;

	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_sig_buf,  input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_err_buf,  input_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_in_err_tmp,  output_node_size * N * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_out_err_buf, output_node_size * frame_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_input_index, output_node_size * N * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_W, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_b, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_W, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_b, output_node_size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_dW, output_node_size * M * N * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_hidden_db, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_dW, output_node_size * M * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_output_db, output_node_size * sizeof(float)));
	
	hipDeviceSynchronize();
	auto time1 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_input_index, input_index, output_node_size * N * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_W, hidden_W, output_node_size * M * N * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_hidden_b, hidden_b, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_W, output_W, output_node_size * M * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_output_b, output_b, output_node_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time2 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(dev_in_sig_buf,  in_sig_buf,  input_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_out_err_buf, out_err_buf, output_node_size * frame_size * sizeof(float), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	auto time3 = std::chrono::system_clock::now();

	bbcu_MicroMlp_backward<N, M>(
			dev_in_sig_buf,
			dev_in_err_buf,
			dev_in_err_tmp,
			dev_out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			dev_input_index,
			dev_hidden_W,
			dev_hidden_b,
			dev_hidden_dW,
			dev_hidden_db,
			dev_output_W,
			dev_output_b,
			dev_output_dW,
			dev_output_db
		);


	hipDeviceSynchronize();
	auto time4 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipMemcpy(in_err_buf, dev_in_err_buf, input_node_size * frame_size * sizeof(float), hipMemcpyDeviceToHost));

	CUDA_SAFE_CALL(hipMemcpy(hidden_dW, dev_hidden_dW, output_node_size * M * N * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(hidden_db, dev_hidden_db, output_node_size * M * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(output_dW, dev_output_dW, output_node_size * M * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(output_db, dev_output_db, output_node_size * sizeof(float), hipMemcpyDeviceToHost));

	hipDeviceSynchronize();
	auto time5 = std::chrono::system_clock::now();

	CUDA_SAFE_CALL(hipFree(dev_in_sig_buf));
	CUDA_SAFE_CALL(hipFree(dev_in_err_buf));
	CUDA_SAFE_CALL(hipFree(dev_in_err_tmp));
	CUDA_SAFE_CALL(hipFree(dev_out_err_buf));
	CUDA_SAFE_CALL(hipFree(dev_input_index));
	CUDA_SAFE_CALL(hipFree(dev_hidden_W));
	CUDA_SAFE_CALL(hipFree(dev_hidden_b));
	CUDA_SAFE_CALL(hipFree(dev_output_W));
	CUDA_SAFE_CALL(hipFree(dev_output_b));
	CUDA_SAFE_CALL(hipFree(dev_hidden_dW));
	CUDA_SAFE_CALL(hipFree(dev_hidden_db));
	CUDA_SAFE_CALL(hipFree(dev_output_dW));
	CUDA_SAFE_CALL(hipFree(dev_output_db));

	hipDeviceSynchronize();
	auto time6 = std::chrono::system_clock::now();

	double elapsed_malloc       = std::chrono::duration_cast<std::chrono::milliseconds>(time1-time0).count();
	double elapsed_cpu_to_gpu_p = std::chrono::duration_cast<std::chrono::milliseconds>(time2-time1).count();
	double elapsed_cpu_to_gpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time3-time2).count();
	double elapsed_kernel       = std::chrono::duration_cast<std::chrono::milliseconds>(time4-time3).count();
	double elapsed_gpu_to_cpu   = std::chrono::duration_cast<std::chrono::milliseconds>(time5-time4).count();
	double elapsed_free         = std::chrono::duration_cast<std::chrono::milliseconds>(time6-time5).count();
//	double kernel_flops = (double)output_node_size *(double) frame_size * (16.0*6.0+16.0+16.0)*2.0 / elapsed_kernel / 1000000.0;
	std::cout << "malloc               : " << elapsed_malloc       << " [msec]" << std::endl;
	std::cout << "param copy(cpu->gpu) : " << elapsed_cpu_to_gpu_p << " [msec]" << std::endl;
	std::cout << "data copy(cpu->gpu)  : " << elapsed_cpu_to_gpu   << " [msec]" << std::endl;
	std::cout << "kernel               : " << elapsed_kernel       << " [msec]" << std::endl;
//	 << kernel_flops << " [GFLOPS])" << std::endl;
	std::cout << "data copy(gpu->cpu)  : " << elapsed_gpu_to_cpu   << " [msec]" << std::endl;
	std::cout << "free                 : " << elapsed_free         << " [msec]" << std::endl;
	
	return 0;
}



int MicroMlp6x16_Backward
		(
			const float*	in_sig_buf,
			float*			in_err_buf,
			float*			out_err_buf,
			int				input_node_size,
			int				output_node_size,
			int				frame_size,
			const int*		input_index,
			const float*	hidden_W,
			const float*	hidden_b,
			float*			hidden_dW,
			float*			hidden_db,
			const float*	output_W,
			const float*	output_b,
			float*			output_dW,
			float*			output_db
		)
{

	return MicroMlp_Backward<6, 16>
		(
			in_sig_buf,
			in_err_buf,
			out_err_buf,
			input_node_size,
			output_node_size,
			frame_size,
			input_index,
			hidden_W,
			hidden_b,
			hidden_dW,
			hidden_db,
			output_W,
			output_b,
			output_dW,
			output_db
		);
}

